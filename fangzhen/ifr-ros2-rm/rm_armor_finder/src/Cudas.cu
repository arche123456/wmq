#include "hip/hip_runtime.h"
#include "rm_armor_finder/Cudas.h"
#include <cstdint>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include <utility>

namespace rm_armor_finder {
    namespace cudas {

        __global__ void getRedFromBayerRG_call(cv::cuda::PtrStepSz<uint8_t> src, cv::cuda::PtrStepSz<uint8_t> dst, int thresh,
                                               float t_r, float t_g1, float t_g2, float t_b) {
            int _i = (int(threadIdx.x) + blockIdx.x * blockDim.x);
            int _j = (int(threadIdx.y) + blockIdx.y * blockDim.y);
            int i = _i * 2;
            int j = _j * 2;
            if (i >= src.cols || j >= src.rows) return;
            uint8_t r = src(j, i), g1 = src(j, i + 1),
                    g2 = src(j + 1, i), b = src(j + 1, i + 1);
            uint8_t max = r;
            if (g1 > max) max = g1;
            if (g2 > max) max = g2;
            if (b > max) max = b;
            float v = t_r * r + t_g1 * g1 + t_g2 * g2 + t_b * b;
            dst(_j, _i) = (v + max) > thresh ? 255 : 0;
        }

        __global__ void getBlueFromBayerRG_call(cv::cuda::PtrStepSz<uint8_t> src, cv::cuda::PtrStepSz<uint8_t> dst, int thresh,
                                                float t_r, float t_g1, float t_g2, float t_b) {
            int _i = (int(threadIdx.x) + blockIdx.x * blockDim.x);
            int _j = (int(threadIdx.y) + blockIdx.y * blockDim.y);
            int i = _i * 2 + 1;
            int j = _j * 2 + 1;
            if (i >= src.cols || j >= src.rows) return;
            uint8_t b = src(j, i), g2 = src(j, i - 1),
                    g1 = src(j - 1, i), r = src(j - 1, i - 1);
            uint8_t max = r;
            if (g1 > max) max = g1;
            if (g2 > max) max = g2;
            if (b > max) max = b;
            float v = t_r * r + t_g1 * g1 + t_g2 * g2 + t_b * b;
            dst(_j, _i) = (v + max) > thresh ? 255 : 0;
        }

        __global__ void toGray_call(cv::cuda::PtrStepSz<uint8_t> src, cv::cuda::PtrStepSz<float> dst, int low, int high) {
            int _i = (int(threadIdx.x) + blockIdx.x * blockDim.x);
            int _j = (int(threadIdx.y) + blockIdx.y * blockDim.y);
            int i = _i * 2;
            int j = _j * 2;
            if (_i >= dst.cols || _j >= dst.rows) return;
            auto v = src(j, i) + src(j + 1, i) + src(j, i + 1) + src(j + 1, i + 1);
            dst(_j, _i) = low <= v && v <= high ? 1.F : 0.F;
        }

        __global__ void getBlueFromBayerRG_2_call(cv::cuda::PtrStepSz<uint8_t> src, cv::cuda::PtrStepSz<uint8_t> dst,
                                                  int h_low, int h_high, int s_low, int s_high, int v_low, int v_high) {
            const int _i = (int(threadIdx.x) + blockIdx.x * blockDim.x);
            const int _j = (int(threadIdx.y) + blockIdx.y * blockDim.y);
            const int i = _i * 2;
            const int j = _j * 2;
            if (_i >= dst.cols || _j >= dst.rows) return;
            const uint8_t r = src(j, i);
            const uint8_t g1 = src(j, i + 1);
            const uint8_t g2 = src(j + 1, i);
            const uint8_t b = src(j + 1, i + 1);
            const uint8_t g = (g1 + g2) / 2;

            uint8_t res = 0;

            const uint8_t m = (g < r ? g : r);         // m = min(r,g,b)
            if (b > m) {                               // V = B
                const int C = b - m;                   // C = V - m
                const int H = (r - g) * 255 / C;       // H = 60° * ((R-G)/C+4) if V = B
                const int S = b == 0 ? 0 : C * 255 / b;// S = C/V
                // H: [-255,255] S: [0,255] V: [0,255]
                if (h_low <= H && H <= h_high && s_low <= S && S <= s_high && v_low <= b && b <= v_high) {
                    res = 255;
                }
            }
            dst(_j, _i) = res;
        }

        __global__ void getRedFromBayerRG_2_call(cv::cuda::PtrStepSz<uint8_t> src, cv::cuda::PtrStepSz<uint8_t> dst,
                                                 int h_low, int h_high, int s_low, int s_high, int v_low, int v_high) {
            const int _i = (int(threadIdx.x) + blockIdx.x * blockDim.x);
            const int _j = (int(threadIdx.y) + blockIdx.y * blockDim.y);
            const int i = _i * 2;
            const int j = _j * 2;
            if (_i >= dst.cols || _j >= dst.rows) return;
            // int b = src(j, i);// swap r and b define: 交换红蓝定义, 避免Red的H计算被切分为两段
            // int g = (src(j, i + 1) + src(j + 1, i)) / 2;
            // int r = src(j + 1, i + 1);
            // uint8_t cnt = 0;

            int r = 0, g = 0, b = 0;
            uint8_t cnt = 0;
            constexpr const int neibor_size = 1;
            for (int x = -2 * neibor_size; x <= 2 * neibor_size; x += 2) {
                if (x + i < 0 || x + i >= src.cols) continue;
                for (int y = -2 * neibor_size; y <= 2 * neibor_size; y += 2) {
                    if (y + j < 0 || y + j >= src.rows) continue;
                    b += src(j + y, i + x);
                    g += (src(j + y, i + x + 1) + src(j + y + 1, i + x)) / 2;
                    r += src(j + y + 1, i + x + 1);
                    cnt += 1;
                }
            }
            r /= cnt;
            g /= cnt;
            b /= cnt;

            uint8_t res = 0;

            const int m = (g < r ? g : r);             // m = min(r,g,b)
            if (b > m) {                               // V = B
                const int C = b - m;                   // C = V - m
                const int H = (r - g) * 255 / C;       // H = 60° * ((R-G)/C+4) if V = B
                const int S = b == 0 ? 0 : C * 255 / b;// S = C/V
                // H: [-255,255] S: [0,255] V: [0,255]
                if (h_low <= H && H <= h_high && s_low <= S && S <= s_high && v_low <= b && b <= v_high) {
                    res = 255;
                }
            }
            dst(_j, _i) = res;
        }

        // __global__ void toDoublePtr_call(cv::cuda::PtrStepSz<uint8_t> src, double *dst) {
        //     int x = blockIdx.x * blockDim.x + threadIdx.x;
        //     int y = blockIdx.y * blockDim.y + threadIdx.y;

        //     if (x < src.cols && y < src.rows) {
        //         dst[y * src.cols + x] = static_cast<double>(src(y, x));
        //     }
        // }

        void toGray(const cv::cuda::GpuMat &src, const cv::cuda::GpuMat &dst, uint8_t low, uint8_t high, cv::cuda::Stream &stream) {
            CV_DbgAssert(src.type() == CV_8UC1);
            CV_DbgAssert(dst.type() == CV_32FC1);
            CV_DbgAssert(!src.empty() && !dst.empty());

            dim3 blockDim(32, 32);
            dim3 gridDim((dst.cols + blockDim.x - 1) / blockDim.x, (dst.rows + blockDim.y - 1) / blockDim.y);
            auto s = cv::cuda::StreamAccessor::getStream(stream);
            toGray_call<<<gridDim, blockDim, 0, s>>>(src, dst, low * 4, high * 4);
        }


        void getColorFromBayerRG(const cv::cuda::GpuMat &src, const cv::cuda::GpuMat &dst, bool getRed,
                                 int h_low, int h_high, int s_low, int s_high, int v_low, int v_high,
                                 cv::cuda::Stream &stream) {
            CV_DbgAssert(src.type() == CV_8UC1);
            CV_DbgAssert(!src.empty() && !dst.empty());

            dim3 blockDim(32, 32);
            dim3 gridDim((dst.cols + blockDim.x - 1) / blockDim.x, (dst.rows + blockDim.y - 1) / blockDim.y);
            auto s = cv::cuda::StreamAccessor::getStream(stream);
            // if (getRed)
            //     getRedFromBayerRG_call<<<gridDim, blockDim, 0, s>>>(src, dst, thresh,
            //                                                         t_r, t_g1, t_g2, t_b);
            // else
            //     getBlueFromBayerRG_call<<<gridDim, blockDim, 0, s>>>(src, dst, thresh,
            //                                                          t_r, t_g1, t_g2, t_b);
            if (getRed)
                getRedFromBayerRG_2_call<<<gridDim, blockDim, 0, s>>>(src, dst, -255, 255, 150, 255, 50, 255);
            else
                getBlueFromBayerRG_2_call<<<gridDim, blockDim, 0, s>>>(src, dst, -255, 255, 150, 255, 50, 255);
        }

        // void toDoublePtr(const cv::cuda::GpuMat &src, double *&dst) {
        //     dim3 blockDim(std::min(src.cols, 32), std::min(src.rows, 32));
        //     dim3 gridDim((src.cols + blockDim.x - 1) / blockDim.x, (src.rows + blockDim.y - 1) / blockDim.y);
        //     toDoublePtr_call<<<gridDim, blockDim>>>(src, dst);
        // }
    }// namespace cudas
}// namespace rm_armor_finder